#include "hip/hip_runtime.h"
#include "quantization.h"
#define THREADS 1024
#define MAX_CENTROIDS 1024

void printVec(float vec[], int n)
{
        printf("[");
        printf("%f",vec[0]);
        for(int i= 1; i<n; i++)
        {
                printf(",%f",vec[i]);
        }
        printf("]\n");
        return;
}


void printVec(int vec[], int n)
{
        printf("[");
        printf("%d",vec[0]);
        for(int i= 1; i<n; i++)
        {
                printf(",%d",vec[i]);
        }
        printf("]\n");
        return;
}

void printPoint3(point3 p)
{
        printf("[%f,%f,%f]\n",p.x,p.y,p.z);
        return;
}

void printPoint3Array(point3 *p, int n)
{
        for(int i=0; i<n; i++)
        {
                printPoint3(p[i]);
        }
}

__device__ point3 addPoint3(point3 p1,point3 p2)
{
        point3 sum;
        sum.x=p1.x+p2.x;
        sum.y=p1.y+p2.y;
        sum.z=p1.z+p2.z;

        return sum;
}

__device__ point3 mulPoint3(point3 p, float s)
{
        point3 mul;
        mul.x=s*p.x;
        mul.y=s*p.y;
        mul.z=s*p.z;

        return mul;
}

point3 randomPoint3()
{
        float x = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
        float y = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
        float z = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);

        point3 randomPoint;
        const float min_z =0.8;
        const float max_z =3;
        const float min_x = -1.5, max_x =1.5;
        const float min_y = -1.5, max_y =1.5;
        randomPoint.x = (max_x-min_x)*x+min_x;
        randomPoint.y = (max_y-min_y)*y+min_y;
        randomPoint.z = (max_z-min_z)*z+min_z;
        return randomPoint;

}

point3 randomPoint3(point3 min, point3 max)
{
        //generates pseudorandom number between 0 and 1
        float x = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
        float y = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
        float z = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);

        // shift and center around the cloud
        point3 randomPoint;
        randomPoint.x = (max.x-min.x)*x+min.x;
        randomPoint.y = (max.y-min.y)*y+min.y;
        randomPoint.z = (max.z-min.z)*z+min.z;
        return randomPoint;

}

void initializeCodebook(point3 * codebook, point3 minPoint,
                        point3 maxPoint,int nClusters)
{
        //loop over all clusters
        printf("Initializing centroids\n");
        printf("Max point:\t"); printPoint3(maxPoint);
        printf("\nMin point:\t"); printPoint3(minPoint);
        printf("\n");
        srand(time(NULL));

        for (unsigned int i = 0; i < nClusters; i++)
        {
                codebook[i] = randomPoint3(minPoint,maxPoint);
                //printPoint3(codebook[i]);
        }
        return;
}

__device__ float euclideanDistance(point3 p1, point3 p2)
{
        return ( (p1.x-p2.x)*(p1.x-p2.x)+(p1.y-p2.y)*(p1.y-p2.y)+(p1.z-p2.z)*(p1.z-p2.z));
}

__device__ void setZerosDevice(point3 *p,int n)
{
        //sets an array of points with only 0
        point3 zero;
        zero.x=0; zero.y=0; zero.z=0;
        for(int i = 0; i < n; i++)
        {
                p[i]=zero;
        }

        return;
}

__device__ void setZerosDevice(int *p,int n)
{
        //sets an array of points with only 0
        int zero = 0;
        for(int i = 0; i < n; i++)
        {
                p[i]=zero;
        }

        return;
}

__global__ void distanceKernel(point3 * points, point3 * centroids,float * distances, int k, int n )
{
        //distances[k*n]
        /*The distance array is a 2d array but i flattened it into a one dimensional array cause i am
           to lazy to create a 2d array on cuda. */
        int idx = threadIdx.x + blockIdx.x * blockDim.x;
        if (idx < n) {

                for(int i= 0; i< k; i++)
                {
                        //distances[point,centroid]
                        distances[n*i+idx]=euclideanDistance(points[idx],centroids[i]);
                        //distance[centroid, point]
                        //distances[i+k*idx]=euclideanDistance(points[idx],centroids[i]);
                }
        }
        return;
}

__global__ void makePartition(int *partition,
                              float *distances, int *histogram,
                              int k, int n)
{
        int idx  = threadIdx.x + blockIdx.x * blockDim.x;
        //__shared__ int sharedPartition[THREADS];
        //setZerosDevice(histogram,k);
        //__syncthreads();
        if (idx < n) {
                //int tid = threadIdx.x;
                int minixd=0;
                float minDist=distances[idx];
                for(int i=1; i<k; i++)
                {
                        if(minDist>distances[idx+n*i])
                        {
                                minixd=i;
                                minDist=distances[idx+n*i];
                        }
                }
                partition[idx]=minixd;
                atomicAdd(&histogram[minixd],1);
        }
        // __syncthreads();
        // if (idx==0) {
        //   printf("-----\n" );
        //         for (int i = 0; i < k; i++) {
        //                 printf("Histogram[%d]=%d\n",i,histogram[i] );
        //         }
        // }
        return;
}

__global__ void prepareReduceArray(point3 *points, int *partition, point3 *reduceArrray, int centroid, int n)
{
        //takes the whole partition array and outputs a single array that only has points belonging to a
        //particular centroid
        int idx = blockIdx.x*blockDim.x+threadIdx.x;
        if(idx < n)
        {
                //set all to 0;
                //copy to shared if assigned to jth centroid
                if(partition[idx]==centroid)
                {
                        //only copy from partition j
                        reduceArrray[idx]=points[idx];
                }
                else
                {
                        point3 zero;
                        zero.x=0; zero.y=0; zero.z=0;
                        reduceArrray[idx]=zero;
                }
        }
        return;
}

__global__ void recalcCentroidsInner(point3 * points,
                                     point3* partialResult,
                                     int *partition,
                                     int n)
{
        //n = histogram[n]
        __shared__ point3 pointsShared[THREADS];
        //__shared__ int partitionShared[THREADS];
        int idx = blockIdx.x*blockDim.x+threadIdx.x;
        if(idx < n)
        {
                int tid = threadIdx.x;
                //set all to 0;
                setZerosDevice(pointsShared,THREADS); //Size of shared memory
                //setZerosDevice(partitionShared,THREADS); //Size of shared memory

                __syncthreads();
                //only copy from partition j
                pointsShared[tid]=points[idx];
                //makes two arrays [k k k 0 0 0 0 k k k]
                //[1 1 1 0 0 0 0 1 1 1]
                /*Array of valid points and zeros and array of ones*/
                __syncthreads();
                for (int s = blockDim.x/2; s>0; s>>=1)
                {
                        if(tid<s)
                        {
                                {
                                        pointsShared[tid]=addPoint3(
                                                pointsShared[tid],
                                                pointsShared[tid+s]);

                                }
                        }
                        __syncthreads();
                }
                if(tid==0) {
                        //printf("Block: %d\n",blockIdx.x);
                        //printf("Number of elements in  %d is %d\n",k, partitionShared[0]);
                        //partitionOut[blockIdx.x] = partitionShared[0];
                        partialResult[blockIdx.x] = pointsShared[0];
                        //centroids[k] = mulPoint3(pointsShared[0],1.0/partitionShared[0]);
                }
        }
        __syncthreads();
}

__global__ void recalcCentroidsOuter(point3 * points,
                                     point3* centroids,
                                     int *partition,
                                     int *histogram,
                                     int k, int n)
{
        //histogram is a count of how many elements belong to each centroid
        __shared__ point3 pointsShared[THREADS];
        int idx = blockIdx.x*blockDim.x+threadIdx.x;
        if(idx < n)
        {
                int tid = threadIdx.x;

                //set all to 0; //El error estaba en alocar n cachos
                setZerosDevice(pointsShared,THREADS); //Size of shared memory
                __syncthreads();

                pointsShared[tid]=points[idx];
                __syncthreads();
                for (int s = blockDim.x/2; s>0; s>>=1)
                {
                        if(tid<s)
                        {
                                {
                                        pointsShared[tid]=addPoint3(
                                                pointsShared[tid],
                                                pointsShared[tid+s]);

                                }
                        }
                        __syncthreads();
                }
                if(tid==0)
                {
                        //printf("Thread: %d\t",idx);
                        //printf("[Inner] Number of elements in  %d is %d\n",k, partitionShared[0]);
                        // printf("Centroid[%d] prior:%f,%f,%f\n",
                        //        k,centroids[k].x,centroids[k].y,centroids[k].z);
                        if(histogram[k]>0) {
                                centroids[k] = mulPoint3(pointsShared[0],1.0/histogram[k]);
                        }
                }
        }
        __syncthreads();
}

__global__ void partitionToLocal(point3 *points,
                                 point3 *partitionPoints,
                                 //point3* centroids,
                                 int *partition,
                                 int *localPartition,
                                 int k, int n)
{
        __shared__ point3 pointsShared[THREADS];
        __shared__ int partitionShared[THREADS];
        int idx = blockIdx.x*blockDim.x+threadIdx.x;
        if(idx < n)
        {
                int tid = threadIdx.x;

                //set all to 0;
                setZerosDevice(pointsShared,THREADS); //Size of shared memory
                setZerosDevice(partitionShared,THREADS); //Size of shared memory
                //copy to shared if assigned to jth centroid
                __syncthreads();
                if(partition[idx]==k)
                {
                        //only copy from partition j
                        pointsShared[tid]=points[idx];
                        partitionShared[tid]=1;
                } //makes two arrays [j j j 0 0 0 0 j j j]
                __syncthreads();

                partitionPoints[idx]=pointsShared[tid];
                localPartition[idx]=partitionShared[tid];
        }
}

void initializeCentroids(point3 *points, point3 aleatorios)
{
        return;
}
//points todos los puntos
//aleatorios vectores aleatorios para perturbar el centroide

void kmeans(point3 *h_points, int *h_partition,
            point3* h_codebook,
            int iterations, int clusters, int nPoints)
{

        printf("Received: %d\n",nPoints);

        //Pointers
        point3 *d_points, *d_codebook, *d_partialSum;
        float *d_distances,*h_distances;
        int *d_partition, *d_partialPart, *d_histogram;
        point3 *d_reduceArray;
        //sizes
        int nPointsSize   = nPoints*sizeof(point3);
        int clustersSize  = clusters*sizeof(point3);
        int distanceSize  = nPoints*clusters*sizeof(float);
        int partitionSize = nPoints*sizeof(int);
        int histogramSize = clusters*sizeof(int);

        h_distances = (float *) malloc(distanceSize);
        //h_partition = (int *) malloc(partitionSize);

        hipMalloc((void**)&d_points,nPointsSize);
        hipMalloc((void**)&d_codebook,clustersSize);
        hipMalloc((void**)&d_distances,distanceSize);
        hipMalloc((void**)&d_partition,partitionSize);
        hipMalloc((void**)&d_partialSum,THREADS*sizeof(point3));
        hipMalloc((void**)&d_partialPart,THREADS*sizeof(int));

        hipMalloc((void**)&d_reduceArray,nPointsSize);
        hipMalloc((void**)&d_histogram,histogramSize);

        hipMemcpy(d_points,h_points,nPointsSize,hipMemcpyHostToDevice);
        hipMemcpy(d_codebook,h_codebook,clustersSize,hipMemcpyHostToDevice);

        int blks = (nPoints + THREADS - 1) / THREADS;
        ///if blks > THREADS return error not enough kenerls
        //int blks = nPoints/ THREADS;
        printf("Issuing %d blocks with %d threads\n",blks, THREADS);
        //hipDeviceSynchronize();
        // recalcCentroids<<<(nPoints + THREADS - 1) / THREADS,THREADS>>>
        // (d_points,d_codebook,d_partition,clusters,nPoints);
        for (int m = 0; m <iterations; m++) {
                distanceKernel<<<blks,THREADS>>>
                (d_points,d_codebook,d_distances,clusters,nPoints);
                //hipDeviceSynchronize();
                hipMemset(d_histogram, 0, histogramSize);
                makePartition<<<blks,THREADS>>>
                (d_partition,d_distances,d_histogram,clusters,nPoints);
                for(int i= 0; i<clusters; i++)
                {
                        prepareReduceArray<<<blks,THREADS>>>
                        (d_points,d_partition,d_reduceArray,i,nPoints);
                        //aqui iria un while
                        recalcCentroidsInner<<<blks,THREADS>>>
                        (d_reduceArray,d_reduceArray,d_partition,nPoints);
                        //Aqui acabaria el while
                        recalcCentroidsOuter<<<1,THREADS>>> //accccesing ilegal meory ?
                        (d_reduceArray,d_codebook,d_partition,d_histogram,i,blks);
                        if (hipPeekAtLastError() != hipSuccess) {
                                printf("kernel launch error: %s\n", hipGetErrorString(hipGetLastError()));
                        }

                }
        }
        hipDeviceSynchronize();
        hipMemcpy(h_distances,d_distances,
                   distanceSize,hipMemcpyDeviceToHost);
        hipMemcpy(h_partition,d_partition,
                   partitionSize,hipMemcpyDeviceToHost);
        hipMemcpy(h_codebook,d_codebook,
                   clustersSize,hipMemcpyDeviceToHost);
        hipMemcpy(h_partition,d_partition,
                   partitionSize,hipMemcpyDeviceToHost);
        //printVec(h_partition,nPoints);
        //printVec(h_distances,nPoints*clusters);
        //printVec(h_partition,nPoints);
        printf("---Optimized centroids---\n");
        printPoint3Array(h_codebook, clusters);
        hipFree(d_points); hipFree(d_distances); hipFree(d_codebook);
        hipFree(d_partition); hipFree(d_partialSum); hipFree(d_partialPart);

        free(h_distances);
        return;
}

void serializeQuantization(point3* points, point3* codebook,
                           int * partition, int n, int k, char *filename)
{
        //points    Elements to be serialized
        //codebook  Final vectors
        //partition Correspondence
        //element points[i] belongs to codebook[partition[i]]
        //n number of points, k number of clusters
        printf("Writing to %s",filename);
        std::ofstream vqOut;
        vqOut.open(filename);
        vqOut<<"Points: " <<n << "\n";
        vqOut<<"Clusters: " <<k << "\n";
        vqOut<<"Codebook:\n";
        for(int i=0; i<k; i++)
        {
                vqOut<<codebook[i].x<<",";
                vqOut<<codebook[i].y<<",";
                vqOut<<codebook[i].z<<"\n";
        }
        vqOut<<"Points:\n";
        for(int i=0; i<n; i++)
        {
                vqOut<<points[i].x<<",";
                vqOut<<points[i].y<<",";
                vqOut<<points[i].z<<",";
                vqOut<<partition[i]<<"\n";
        }
        vqOut.close();
        return;

}


__global__ void reducePoint3(point3 *cloud_in, point3 *cloud_out)
{
        __shared__ point3 pointsShared[THREADS];
        int idx = blockIdx.x*blockDim.x+threadIdx.x;
        int tid = threadIdx.x;
        pointsShared[tid]=cloud_in[idx];
        __syncthreads();

        for (int s = blockDim.x/2; s>0; s>>=1)
        {
                if(tid<s)
                {
                        pointsShared[tid]=addPoint3(
                                pointsShared[tid],
                                pointsShared[tid+s]);

                }
                __syncthreads();
        }


        if(tid==0) cloud_out[blockIdx.x]=pointsShared[0];
}

int getFreeMem()
{
        size_t *libre=NULL, *ocupada=NULL; //sí se me acabo el ingles
        hipMemGetInfo(libre,ocupada);
        return *libre;
}
